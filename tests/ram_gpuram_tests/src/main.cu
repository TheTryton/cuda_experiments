#include <cstdint>
#include <vector>
#include <chrono>
#include <cassert>
#include <iostream>
#include <format>

using namespace std;

double measure_ram_ram_transfer_speed(const vector<uint8_t> & src, vector<uint8_t> & dst, size_t iterations)
{
    assert(src.size() == dst.size());

    using seconds = chrono::duration<double, ratio<1, 1>>;

    const auto start = chrono::high_resolution_clock::now();
    for (size_t i = 0; i < iterations; ++i)
    {
        memcpy(data(dst), data(src), sizeof(uint8_t) * size(src));
    }
    const auto stop = chrono::high_resolution_clock::now();

    const auto bytesTransferred = sizeof(uint8_t) * size(src);
    const auto timeTakenSeconds = chrono::duration_cast<seconds>(stop - start).count();

    const auto bytesPerSecond = static_cast<double>(bytesTransferred) / timeTakenSeconds * static_cast<double>(iterations);
    return bytesPerSecond;
}

double measure_gpuram_gpuram_transfer_speed(const vector<uint8_t> & src,size_t iterations)
{
    assert(src.size() == dst.size());

    uint8_t * srcDevice;
    hipMalloc(&srcDevice, sizeof(uint8_t) * size(src));
    uint8_t * dstDevice;
    hipMalloc(&dstDevice, sizeof(uint8_t) * size(src));

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (size_t i = 0; i < iterations; ++i)
    {
        hipMemcpyAsync(dstDevice, srcDevice, sizeof(uint8_t) * size(src), hipMemcpyDeviceToDevice);
    }
    hipEventRecord(stop);

    hipStreamSynchronize(0);

    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    const auto bytesTransferred = sizeof(uint8_t) * size(src);
    const auto timeTakenSeconds = static_cast<double>(milliseconds) / 1000;

    const auto bytesPerSecond = static_cast<double>(bytesTransferred) / timeTakenSeconds * static_cast<double>(iterations);
    return bytesPerSecond;
}

double measure_ram_gpuram_transfer_speed(const vector<uint8_t> & src, size_t iterations)
{
    assert(src.size() == dst.size());

    uint8_t * srcHost;
    hipHostAlloc(&srcHost, sizeof(uint8_t) * size(src), hipHostMallocWriteCombined);
    std::memcpy(srcHost, data(src), sizeof(uint8_t) * size(src));
    uint8_t * dstDevice;
    hipMalloc(&dstDevice, sizeof(uint8_t) * size(src));

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (size_t i = 0; i < iterations; ++i)
    {
        hipMemcpyAsync(dstDevice, srcHost, sizeof(uint8_t) * size(src), hipMemcpyHostToDevice);
    }
    hipEventRecord(stop);

    hipStreamSynchronize(0);

    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    const auto bytesTransferred = sizeof(uint8_t) * size(src);
    const auto timeTakenSeconds = static_cast<double>(milliseconds) / 1000;

    const auto bytesPerSecond = static_cast<double>(bytesTransferred) / timeTakenSeconds * static_cast<double>(iterations);
    return bytesPerSecond;
}

double measure_gpuram_ram_transfer_speed(const vector<uint8_t> & src, size_t iterations)
{
    assert(src.size() == dst.size());

    uint8_t * srcDevice;
    hipMalloc(&srcDevice, sizeof(uint8_t) * size(src));
    uint8_t * dstHost;
    hipHostAlloc(&dstHost, sizeof(uint8_t) * size(src), hipHostMallocDefault);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (size_t i = 0; i < iterations; ++i)
    {
        hipMemcpyAsync(srcDevice, dstHost, sizeof(uint8_t) * size(src), hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop);

    hipStreamSynchronize(0);

    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    const auto bytesTransferred = sizeof(uint8_t) * size(src);
    const auto timeTakenSeconds = static_cast<double>(milliseconds) / 1000;

    const auto bytesPerSecond = static_cast<double>(bytesTransferred) / timeTakenSeconds * static_cast<double>(iterations);
    return bytesPerSecond;
}

int main()
{
    constexpr size_t bytesCount = 4ull * 1024 * 1024 * 1024;
    constexpr size_t iterationCount = 10;

    vector<uint8_t> src(bytesCount);
    vector<uint8_t> dst(bytesCount);

    const auto transferSpeedRamRam = measure_ram_ram_transfer_speed(src, dst, iterationCount);
    const auto transferSpeedGpuRamGpuRam = measure_gpuram_gpuram_transfer_speed(src, iterationCount);
    const auto transferSpeedRamGpuRam = measure_gpuram_ram_transfer_speed(src, iterationCount);
    const auto transferSpeedGpuRamRam = measure_ram_gpuram_transfer_speed(src, iterationCount);

    cout << format("[CPU->CPU] Transfer speed {} GB/s", transferSpeedRamRam / 1024 / 1024 / 1024) << endl;
    cout << format("[GPU->GPU] Transfer speed {} GB/s", transferSpeedGpuRamGpuRam / 1024 / 1024 / 1024) << endl;
    cout << format("[CPU->GPU] Transfer speed {} GB/s", transferSpeedRamGpuRam / 1024 / 1024 / 1024) << endl;
    cout << format("[GPU->CPU] Transfer speed {} GB/s", transferSpeedGpuRamRam / 1024 / 1024 / 1024) << endl;

    return 0;
}

